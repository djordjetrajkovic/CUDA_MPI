#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__
void kmultiply(const float* a, float* b, int n) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < n)
        b[i] *= a[i];
}

extern "C"
void launch_multiply(const float* a, float* b, int n) {
    float* dA;
    float* dB;
    int cerr;

    cerr = hipMalloc((void**)&dA, n*sizeof(float));
    cerr = hipMalloc((void**)&dB, n*sizeof(float));
    cerr = hipMemcpy(dA, a, n*sizeof(float), hipMemcpyHostToDevice);
    cerr = hipMemcpy(dB, b, n*sizeof(float), hipMemcpyHostToDevice);

    kmultiply<<<ceil((float)n/256), 256>>>(dA, dB, n);

    cerr = hipDeviceSynchronize();

    cerr = hipMemcpy(b, dB, n*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
}
